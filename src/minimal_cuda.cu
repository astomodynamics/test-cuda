#include <iostream>
#include <hip/hip_runtime.h>

// CUDA kernel function to add two vectors
__global__ void vectorAdd(const float* A, const float* B, float* C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    int N = 1 << 20; // 1 million elements

    // Allocate host memory
    float* h_A = new float[N];
    float* h_B = new float[N];
    float* h_C = new float[N];

    // Initialize host arrays
    for (int i = 0; i < N; i++) {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(i * 2);
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, N * sizeof(float));
    hipMalloc((void**)&d_B, N * sizeof(float));
    hipMalloc((void**)&d_C, N * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_A, h_A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * sizeof(float), hipMemcpyHostToDevice);

    // Set the block size and grid size
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;

    // Launch the kernel
    vectorAdd<<<numBlocks, blockSize>>>(d_A, d_B, d_C, N);

    // Copy result back to host
    hipMemcpy(h_C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);

    // Verify the result
    for (int i = 0; i < 10; i++) {
        std::cout << "C[" << i << "] = " << h_C[i] << std::endl;
    }

    // Free device and host memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}
